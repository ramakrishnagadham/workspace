#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>
#include <iostream>

#define N 40000000 // 40 million

int main() {
    int *h_data, *d_data;
    size_t size = N * sizeof(int);

    // Allocate host memory
    h_data = (int*)malloc(size);
    
    // Initialize host array with data
    for (int i = 0; i < N; ++i) {
        h_data[i] = /* your data initialization */;
    }

    // Allocate device memory
    hipMalloc(&d_data, size);
    
    // Copy data to device
    hipMemcpy(d_data, h_data, size, hipMemcpyHostToDevice);

    // Allocate temporary storage
    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_data, d_data, N);
    hipMalloc(&d_temp_storage, temp_storage_bytes);

    // Sort data on the device
    hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_data, d_data, N);

    // Copy sorted data back to host
    hipMemcpy(h_data, d_data, size, hipMemcpyDeviceToHost);

    // Free memory
    hipFree(d_data);
    hipFree(d_temp_storage);
    free(h_data);

    return 0;
}
